
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 1025
#define THREADBLOCKSIZE 256

__global__ void addVector(int vectorA[SIZE],int vectorB[SIZE],int vectorC[SIZE]);

int main(int argc, char const *argv[])
{
    int vectorA[SIZE];
    int vectorB[SIZE];
    int vectorC[SIZE];

    for (int i = 0; i < SIZE; i++)
    {
        vectorA[i] = i;
        vectorB[i] = SIZE-i;
    }

    int *vectorA_cuda;
    int *vectorB_cuda;
    int *vectorC_cuda;
    hipError_t code;
    hipMalloc((void **)&vectorA_cuda,sizeof(int)*SIZE);
    code = hipGetLastError();

    if (code != hipSuccess)
    {
        printf("Error occured in %s at function %s line no %d \n",__FILE__,__FUNCTION__,__LINE__);
        exit(1);
    }
    hipMalloc((void **)&vectorB_cuda,sizeof(int)*SIZE);
    code = hipGetLastError();
    if (code != hipSuccess)
    {
        printf("Error occured in %s at function %s line no %d \n",__FILE__,__FUNCTION__,__LINE__);
        exit(1);
    }
    hipMalloc((void **)&vectorC_cuda,sizeof(int)*SIZE);
    code = hipGetLastError();
    if (code != hipSuccess)
    {
        printf("Error occured in %s at function %s line no %d \n",__FILE__,__FUNCTION__,__LINE__);
        exit(1);
    }
    
    hipMemcpy(vectorA_cuda,vectorA,sizeof(int)*SIZE,hipMemcpyHostToDevice);
    code = hipGetLastError();
    if (code != hipSuccess)
    {
        printf("Error occured in %s at function %s line no %d \n",__FILE__,__FUNCTION__,__LINE__);
        exit(1);
    }
    hipMemcpy(vectorB_cuda,vectorB,sizeof(int)*SIZE,hipMemcpyHostToDevice);
    code = hipGetLastError();
    if (code != hipSuccess)
    {
        printf("Error occured in %s at function %s line no %d \n",__FILE__,__FUNCTION__,__LINE__);
        exit(1);
    }

    int noOfThreadBlocks = (int)(SIZE/THREADBLOCKSIZE) + 1;
    addVector<<<noOfThreadBlocks,THREADBLOCKSIZE>>>(vectorA_cuda,vectorB_cuda,vectorC_cuda);

    hipDeviceSynchronize();
    code = hipGetLastError();
    if (code != hipSuccess)
    {
        printf("Error occured in %s at function %s line no %d \n",__FILE__,__FUNCTION__,__LINE__);
        exit(1);
    }
    hipMemcpy(vectorC,vectorC_cuda,sizeof(int)*SIZE,hipMemcpyDeviceToHost);
    code = hipGetLastError();
    if (code != hipSuccess)
    {
        printf("Error occured in %s at function %s line no %d \n",__FILE__,__FUNCTION__,__LINE__);
        exit(1);
    }
    
    printf("Answer is : ");
    for (int i = 0; i < SIZE; i++)
    {
       printf("%d ",vectorC[i]);
    }
    
    return 0;
}

__global__ void addVector(int vectorA[SIZE],int vectorB[SIZE],int vectorC[SIZE]){
    int thread_Id =blockDim.x*blockIdx.x + threadIdx.x;
    if (thread_Id < SIZE)
        vectorC[thread_Id] = vectorA[thread_Id]+ vectorB[thread_Id];
}
